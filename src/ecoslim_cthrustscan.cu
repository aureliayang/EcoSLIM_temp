#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
extern "C" {
    void scan_int_wrapper( int *data_in, int N, int *data_out)
    {
    thrust::device_ptr<int> dev_ptr_in(data_in);
    thrust::device_ptr<int> dev_ptr_out(data_out);
    thrust::inclusive_scan(dev_ptr_in, dev_ptr_in+N, dev_ptr_out);
    }
    void scan_float_wrapper( float *data_in, int N, float *data_out)
    {
    thrust::device_ptr<float> dev_ptr_in(data_in);
    thrust::device_ptr<float> dev_ptr_out(data_out);
    thrust::inclusive_scan(dev_ptr_in, dev_ptr_in+N, dev_ptr_out);
    }
    void scan_double_wrapper( double *data_in, int N, double *data_out)
    {
    thrust::device_ptr<double> dev_ptr_in(data_in);
    thrust::device_ptr<double> dev_ptr_out(data_out);
    thrust::inclusive_scan(dev_ptr_in, dev_ptr_in+N, dev_ptr_out);
    }
    void scan_longint_wrapper( long long int *data_in, int N, int *data_out)
    {
    thrust::device_ptr<long long int> dev_ptr_in(data_in);
    thrust::device_ptr<int> dev_ptr_out(data_out);
    thrust::inclusive_scan(dev_ptr_in, dev_ptr_in+N, dev_ptr_out);
    }
}